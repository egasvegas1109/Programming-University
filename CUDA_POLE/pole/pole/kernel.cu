#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <>
#include <stdio.h>
#include <cstring>

#define block_size 16
#define kmax 200 //����� �������

__global__ void Kernel(float* uDev, int n, int m) {
	int x = blockIdx.x * blockDim.x + threadIdx.x; //������ ����� �� X
	int y = blockIdx.y * blockDim.y + threadIdx.y; //������ ����� �� Y

	//������� ��� ��������
	int i = threadIdx.x + 1;
	int j = threadIdx.y + 1;

	//�������� ������ ��� �������������� �����
	__shared__ float subPoly[block_size + 2][block_size + 2];

	//���� ��� ��� ��� �����, �� �������
	if (x<0 || x>n - 1 || y<0 || y>m - 1)
		return;

	//��������� ���� ����� � ����������� ������
	subPoly[i][j] = uDev[y * n + x];

	//���� �� �� �������, �� �������
	if (x == 0 || x == n - 1 || y == 0 || y == m - 1)
		return;

	//���� � �������� �� ����� ������� �����, �� �������� ����� ������������ ����� � ����������� ������
	if (threadIdx.x == 0) subPoly[i - 1][j] = uDev[y * n + x - 1];

	//���� � �������� �� ������� ������� �����, �� �������� ������� ������������ ����� � ����������� ������
	if (threadIdx.y == 0) subPoly[i][j - 1] = uDev[(y - 1) * n + x];

	//���� � �������� �� ������ ������� �����, �� �������� ������ ������������ ����� � ����������� ������
	if (threadIdx.x == blockDim.x - 1) subPoly[i + 1][j] = uDev[y * n + x + 1];

	//���� � �������� �� ������ ������� �����, ���������� ������ ������������ ����� � ����������� ������
	if (threadIdx.y == blockDim.y - 1) subPoly[i][j + 1] = uDev[(y + 1) * n + x];

	//�������������� ����� � �����
	__syncthreads();

	//����������� ���� ����� � ���������� �� � �����������, � ����� � � ���������� ������
	subPoly[i][j] = 0.25 * (subPoly[i - 1][j] + subPoly[i + 1][j] + subPoly[i][j - 1] + subPoly[i][j + 1]);
	uDev[y * n + x] = subPoly[i][j];
}

int main(int argc, char** argv) {
	float lx = 1.0, ly = 1.0, h = 0.2;//��������� �������� �� ������� ������

	int n = (int)(lx / h) + 1; //����� ����� �� X
	int m = (int)(ly / h) + 1; //����� ����� �� Y

	float** uHost = new float* [n]; //��������� ������������� ������ U, ������� ����� ����� ������������ � ����������, ��� n - ���������� �������
	for (int i = 0; i < n; i++)
		uHost[i] = new float[m];//m - ���-�� ��������

	float* u = new float[n * m]; //������ U �� �����
	float* uDev = NULL; //������ U �� �������
	int size = n * m * sizeof(float); //������ ������� � ������

	//��������� ��������� ������ ������
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			uHost[i][j] = 0.0;

	//������� � ������ �������
	for (int i = 0; i < n; i++) {
		float x = (i * h);//������������ x � ������ �����

		uHost[i][0] = 60.0 * x * (1.0 - x * x); //AD
		uHost[i][m - 1] = 50.0 * (1.0 - x); //BC
	}

	//������ � ����� �������
	for (int i = 0; i < m; i++) {
		float y = (i * h);

		uHost[0][i] = 50.0 * y * y; //AB
		uHost[n - 1][i] = 0; //CD
	}

	//���������� ������� � ����������
	for (int i = 0; i < n; i++)
		for (int j = 0; j < m; j++)
			u[j * m + i] = uHost[i][j];

	float time = 0.0;
	hipEvent_t tn, tk;
	hipEventCreate(&tn);
	hipEventCreate(&tk);
	hipEventRecord(tn, 0);

	hipMalloc((void**)&uDev, size); //�������� ������ �� �������

	hipMemcpy(uDev, u, size, hipMemcpyHostToDevice);

	for (int k = 0; k < kmax; k++) {
		Kernel << < dim3(n / block_size + 1, m / block_size + 1), dim3(block_size, block_size) >> > (uDev, n, m);

		//���� ���� ��� ��������
		hipDeviceSynchronize();
	}

	hipMemcpy(u, uDev, size, hipMemcpyDeviceToHost);

	hipEventRecord(tk, 0);
	hipEventSynchronize(tk);
	hipEventElapsedTime(&time, tn, tk);

	printf("\nCuda time = %f\n", time / 1000.0);

	const char* error = hipGetErrorString(hipGetLastError());
	printf("Cuda error = %s\n", error);

	FILE* f;
	f = fopen("Rez.txt", "w");

	for (int i = n - 1; i >= 0; i--) {
		for (int j = 0; j < m; j++)
			fprintf(f, "%f   ", u[i * m + j]);
		fprintf(f, "\n");
	}

	fclose(f);

	hipFree(uDev);

	for (int i = 0; i < n; i++)
	{
		delete[] uHost[i];
	}
	delete[] uHost;

	return 0;
}